
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/Im2Col.cu"
#else

static inline void THNN_(Im2Col_shapeCheck)(
                         THCState *state,
                         THCTensor *input,
                         THCTensor *gradOutput,
                         int kH, int kW, int dH, int dW,
                         int padH, int padW, int sH, int sW) {
  THArgCheck(kW > 0 && kH > 0, 9,
             "kernel size should be greater than zero, but got kH: %d kW: %d", kH, kW);
  THArgCheck(sW > 0 && sH > 0, 11,
             "stride should be greater than zero, but got sH: %d sW: %d", sH, sW);
  THArgCheck(dW > 0 && dH > 0, 11,
             "dilation should be greater than zero, but got dH: %d dW: %d", dH, dW);

  int ndim = input->nDimension;
  int dimf = 0;
  int dimh = 1;
  int dimw = 2;

  THCUNN_argCheck(state, ndim == 3, 2, input,
                  "3D input tensor expected but got: %s");

  long nInputPlane  = input->size[dimf];
  long inputHeight  = input->size[dimh];
  long inputWidth   = input->size[dimw];
  long outputHeight = (inputHeight + 2*padH - kH - ((kH - 1)*(dH - 1))) / sH + 1;
  long outputWidth  = (inputWidth + 2*padW - kW - ((kW - 1)*(dW - 1))) / sW + 1;
  long nOutputPlane = nInputPlane * kW * kH;
  long outputLength = outputHeight * outputWidth;

  if (outputWidth < 1 || outputHeight < 1)
      THError("Given input size: (%d x %d x %d). "
              "Calculated output size: (%d x %d). Output size is too small",
              nInputPlane,inputHeight,inputWidth,nOutputPlane,outputLength);

  if (gradOutput != NULL) {
    THCUNN_check_dim_size(state, gradOutput, ndim, 0, nOutputPlane);
    THCUNN_check_dim_size(state, gradOutput, ndim, 1, outputLength);
  }
}

void THNN_(Im2Col_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *output,
           int kW, int kH,
           int dW, int dH,
           int padW, int padH,
           int sW, int sH) {

  THCUNN_assertSameGPU(state, 2, input, output);

  // Params:
  int dimf = 0;
  int dimh = 1;
  int dimw = 2;
  long inputHeight  = input->size[dimh];
  long inputWidth   = input->size[dimw];
  long nInputPlane = input->size[dimf];
  long outputHeight = (inputHeight + 2*padH - (dH * (kH - 1)) + 1) / sH + 1;
  long outputWidth  = (inputWidth + 2*padW - (dW * (kW - 1)) + 1) / sW + 1;
  long nOutputPlane = nInputPlane * kW * kH;
  long outputLength = outputHeight*outputWidth;

  THNN_(Im2Col_shapeCheck)
       (state, input, NULL, kH, kW, dH, dW, padH, padW, sH, sW);

  input = THCTensor_(newContiguous)(state, input);

  // Resize output
  THCTensor_(resize2d)(state, output, nOutputPlane, outputLength);

  // Extract columns:
  im2col(
    THCState_getCurrentStream(state),
    THCTensor_(data)(state, input),
    nInputPlane, inputHeight, inputWidth, kH, kW, padH, padW, sH, sW,
    dH, dW, THCTensor_(data)(state, output)
  );

  THCTensor_(free)(state, input);
}

void THNN_(Im2Col_updateGradInput)(
           THCState *state,
           THCTensor *input,
           THCTensor *gradOutput,
           THCTensor *gradInput,
           int kW, int kH,
           int dW, int dH,
           int padW, int padH,
           int sW, int sH) {

  THCUNN_assertSameGPU(state, 3, input, gradOutput, gradInput);

  // Params
  long inputHeight  = input->size[1];
  long inputWidth   = input->size[2];
  long nInputPlane = input->size[0];

  input = THCTensor_(newContiguous)(state, input);
  gradOutput = THCTensor_(newContiguous)(state, gradOutput);

  THNN_(Im2Col_shapeCheck)
       (state, input, gradOutput, kH, kW, dH, dW, padH, padW, sH, sW);

  // Resize output
  THCTensor_(resize3d)(state, gradInput, nInputPlane, inputHeight, inputWidth);

  // Unpack columns back into input:
  col2im<real, accreal>(
    THCState_getCurrentStream(state),
    THCTensor_(data)(state, gradOutput),
    nInputPlane, inputHeight, inputWidth, kH, kW, padH, padW, sH, sW,
    dH, dW, THCTensor_(data)(state, gradInput)
  );

  THCTensor_(free)(state, input);
  THCTensor_(free)(state, gradOutput);
}


#endif
